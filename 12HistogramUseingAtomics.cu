#include "hip/hip_runtime.h"
// Name: Phil Alcorn
// Histogram useing atomics in global memory and shared memory.
// nvcc 12HistogramUseingAtomics.cu -o temp

/*
 What to do:
 This code generates a series of random numbers and places them into bins based on size ranges using the CPU.

 Your task:
 - Create a binning scheme that utilizes the GPU.
 - Take advantage of both global and shared memory atomic operations.
 - The function call has already been provided.
 - Set the block size to **twice** the number of multiprocessors on the GPU.
*/

/*
 Purpose:
 To learn how to use atomic operations at both the shared and global memory levels.
 Along the way, you'll also learn a bit about generating random numbers using `srand`,
 which will come in handy when we use `hiprand` in a later assignment.
*/

// Include files
#include <sys/time.h>
#include <stdio.h>

/*
 Note: The Max int value is 2,147,483,647
 The length of the sequence of random number that srand generates is 2^32,
 that is 4,294,967,296 which is bigger than the largest int but the max for an unsigned int.
*/

// Defines
#define NUMBER_OF_RANDOM_NUMBERS 2147483
#define NUMBER_OF_BINS 10
#define MAX_RANDOM_NUMBER 100.0f

// Global variables
float *RandomNumbersGPU;
int *HistogramGPU;
float *RandomNumbersCPU;
int *HistogramCPU;
int *HistogramCPUTemp; // Use it to hod the GPU histogram past back so we can compair to CPU histogram.
dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid

hipDeviceProp_t prop;

//Function prototypes
void cudaErrorCheck(const char *, int);
void SetUpCudaDevices();
void AllocateMemory();
void Innitialize();
void CleanUp();
void fillHistogramCPU();
__global__ void fillHistogramGPU(float *, int *);
int main();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

//This will be the layout of the parallel space we will be using.
void SetUpCudaDevices()
{
	hipGetDeviceProperties(&prop, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	
	BlockSize.x = prop.maxThreadDim[0];
	BlockSize.y = 1;
	BlockSize.z = 1;


	// Each multiprocessor handles one block at a time, regardless of the thread
	// count. However, if we make the number of blocks twice as big as the 
	// number of SMs, then our GPU scheduler has some more flexibility. 
	// It can limit how many blocks are waiting on memory and therefore speed
	// up throughput. That is, as soon as one block is finished accessing the
	// memory, then there is another block that's already fetched its memory
	// and is ready to go.
	GridSize.x = 2*prop.multiProcessorCount;
	GridSize.y = 1;
	GridSize.z = 1;
}

//Sets memory on the GPU and CPU for our use.
void AllocateMemory()
{					
	//Allocate Device (GPU) Memory
	hipMalloc(&RandomNumbersGPU, NUMBER_OF_RANDOM_NUMBERS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&HistogramGPU, NUMBER_OF_BINS*sizeof(int));
	cudaErrorCheck(__FILE__, __LINE__);

	//Allocate Host (CPU) Memory
	RandomNumbersCPU = (float*)malloc(NUMBER_OF_RANDOM_NUMBERS*sizeof(float));
	HistogramCPU = (int*)malloc(NUMBER_OF_BINS*sizeof(int));
	HistogramCPUTemp = (int*)malloc(NUMBER_OF_BINS*sizeof(int));
	
	//Setting the the histograms to zero.
	hipMemset(HistogramGPU, 0, NUMBER_OF_BINS*sizeof(int));
	cudaErrorCheck(__FILE__, __LINE__);
	memset(HistogramCPU, 0, NUMBER_OF_BINS*sizeof(int));
}

//Loading random numbers.
void Innitialize()
{
	time_t t;
	srand((unsigned) time(&t));
	
	// rand() returns an int in [0, RAND_MAX] "end points included".
	
	for(int i = 0; i < NUMBER_OF_RANDOM_NUMBERS; i++)
	{		
		RandomNumbersCPU[i] = MAX_RANDOM_NUMBER*(float)rand()/RAND_MAX;	
	}

}

//Cleaning up memory after we are finished.
void CleanUp()
{
	hipFree(RandomNumbersGPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(HistogramGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	free(RandomNumbersCPU); 
	free(HistogramCPU);
	free(HistogramCPUTemp);
	//printf("\n Cleanup Done.");
}

void fillHistogramCPU()
{
	float breakPoint;
	int k, done;
	float stepSize = MAX_RANDOM_NUMBER/(float)NUMBER_OF_BINS;
	
	for(int i = 0; i < NUMBER_OF_RANDOM_NUMBERS; i++)
	{
		breakPoint = stepSize;
		k = 0;
		done =0;
		while(done == 0)
		{
			if(RandomNumbersCPU[i] < breakPoint)
			{
				HistogramCPU[k]++; 
				done = 1;
			}
			
			if(NUMBER_OF_BINS < k)
			{
				printf("\n k is too big\n");
				exit(0);
			}
			k++;
			breakPoint += stepSize;
		}
	}
}

//This is the kernel. It is the function that will run on the GPU.
__global__ void fillHistogramGPU(float *randomNumbers, int *hist)
{

	__shared__ unsigned int local_histogram[NUMBER_OF_BINS];

	int gid = threadIdx.x + blockDim.x * blockIdx.x;	
	int lid = threadIdx.x;
	int total_threads = gridDim.x*blockDim.x;

	// Initialize local hist to 0
	for (int i = lid; i < NUMBER_OF_BINS; i+= blockDim.x) 
	{
		localHist[i] = 0;
	}

	__syncthreads();

	// Looping through all elements in case our vector is larger than 
	// our number of elements. 
	for (int i = gid; i<NUMBER_OF_RANDOM_NUMBERS; i += total_threads)
	{
		// Scale the number to [0,1) then multiply by number of bins, 
		// mapping our number to [0, NUMBER_OF_BINS). Casting to int then
		// gets the right bin.
		int bin = (int) ((RandomNumbersGPU[i] / MAX_RANDOM_NUMBER) * NUMBER_OF_BINS);
		
		// Safeguard against exceeding bin count (shouldn't ever happen)
		if (bin>= NUMBER_OF_BINS) bin = NUMBER_OF_BINS-1; 

		// Add one to the corresponding bin.
		atomicAdd(&local_histogram[bin], 1);
	}
	

	// Add them to the global histogram upon completion
	for (int i = lid; i < NUMBER_OF_BINS; i += blockDim.x) 
	{
		atomicAdd (HistogramGPU[i], localHist[i]);
	}

}

int main()
{
	float time;
	timeval start, end;
	
	long int test = NUMBER_OF_RANDOM_NUMBERS;
	if(2147483647 < test)
	{
		printf("\nThe length of your vector is longer than the largest integer value allowed of 2,147,483,647.\n");
		printf("You should check your code.\n Good Bye\n");
		exit(0);
	}
	
	//Set the thread structure that you will be using on the GPU	
	SetUpCudaDevices();

	//Partitioning off the memory that you will be using and padding with zero vector will be a factor of block size.
	AllocateMemory();

	//Loading up values to be added.
	Innitialize();
	
	gettimeofday(&start, NULL);
	fillHistogramCPU();
	gettimeofday(&end, NULL);
	time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	printf("\nTime on CPU = %.15f milliseconds\n", (time/1000.0));
	
	gettimeofday(&start, NULL);
	//Copy Memory from CPU to GPU		
	hipMemcpyAsync(RandomNumbersGPU, RandomNumbersCPU, NUMBER_OF_RANDOM_NUMBERS*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	fillHistogramGPU<<<GridSize,BlockSize>>>(RandomNumbersGPU, HistogramGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	//Copy Memory from GPU to CPU	
	hipMemcpyAsync(HistogramCPUTemp, HistogramGPU, NUMBER_OF_BINS*sizeof(int), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	gettimeofday(&end, NULL);
	time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	printf("\nTime on GPU = %.15f milliseconds\n", (time/1000.0));
	
	//Check
	for(int i = 0; i < NUMBER_OF_BINS; i++)
	{
		printf("\n Deference in histogram bins %d is %d.", i, abs(HistogramCPUTemp[i] - HistogramCPU[i]));
	}
	
	//You're done so cleanup your mess.
	CleanUp();	
	
	printf("\n\n");
	return(0);
}
